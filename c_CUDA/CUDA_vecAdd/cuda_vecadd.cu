/*
    A basic CUDA demonstration. Two random vectors are added together
    in serial and using a GPU accelerator.

    To compile, use:
        make

    NOTE: CUDA must be installed/loaded before running make. Also, the
          Makefile will probably have to be customized for your system.

    To run, use for example:
        ./cuda_vecadd 100000000
*/
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>


// Add two vectors in serial
void vecAdd(double *h_A, double *h_B, double *h_C, int n)
{
    for( int i = 0; i < n; i++ )
        h_C[i] = h_A[i] + h_B[i];
}


// The CUDA vector addition kernel
__global__
void cudaVecAddKernel( double* A, double* B, double* D, int n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i<n) D[i] = A[i] + B[i];
}


// Add two vectors in CUDA
void cuda_vecAdd(double *h_A, double *h_B, double *h_D, int n)
{
    int size = n * sizeof(double);
    double *d_A, *d_B, *d_D;
    hipError_t err1 = hipSuccess;
    hipError_t err2 = hipSuccess;
    hipError_t err3 = hipSuccess;

    // Allocate memory on the GPU
    err1 = hipMalloc((void **) &d_A, size);
    err2 = hipMalloc((void **) &d_B, size);
    err3 = hipMalloc((void **) &d_D, size);

    if(err1 != hipSuccess) {
        printf("\n%s in %s at line %d\n", hipGetErrorString(err1), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }
    if(err2 != hipSuccess) {
        printf("\n%s in %s at line %d\n", hipGetErrorString(err2), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }
    if(err3 != hipSuccess) {
        printf("\n%s in %s at line %d\n", hipGetErrorString(err3), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    // Copy the data to the GPU
    err1 = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    err2 = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    if(err1 != hipSuccess) {
        printf("\n%s in %s at line %d\n", hipGetErrorString(err1), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }
    if(err2 != hipSuccess) {
        printf("\n%s in %s at line %d\n", hipGetErrorString(err2), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    // CUDA kernel
    int threads = 1024;                         // Threads per block
    int blocks = (n + threads - 1) / threads;   // Blocks per grid
    printf("\n  CUDA kernel was launched with %d blocks of %d threads...", blocks, threads);
    cudaVecAddKernel<<<blocks, threads>>>(d_A, d_B, d_D, n);

    err1 = hipGetLastError();
    if(err1 != hipSuccess) {
        printf("\n%s in %s at line %d\n", hipGetErrorString(err1), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    // Copy the results back to the host
    err1 = hipMemcpy(h_D, d_D, size, hipMemcpyDeviceToHost);

    if(err1 != hipSuccess) {
        printf("\n%s in %s at line %d\n", hipGetErrorString(err1), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    // Deallocate memory on the GPU
    err1 = hipFree(d_A);
    err2 = hipFree(d_B);
    err3 = hipFree(d_D);

    if(err1 != hipSuccess) {
        printf("\n%s in %s at line %d\n", hipGetErrorString(err1), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }
    if(err2 != hipSuccess) {
        printf("\n%s in %s at line %d\n", hipGetErrorString(err2), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }
    if(err3 != hipSuccess) {
        printf("\n%s in %s at line %d\n", hipGetErrorString(err3), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }
}


// Timer
double getTime()
{
    struct timeval TV;
    struct timezone TZ;

    const int RC = gettimeofday(&TV, &TZ);

    return( ((double)TV.tv_sec) + ((double)TV.tv_usec)*1.0e-6 );

}


// Main program
int main (int argc, char** argv)
{
    unsigned int n, i;
    double t0, t1, t2, t3;
    
    n = atoi(argv[1]);
    printf("\nn = %d", n);

    double * A = (double*) malloc( n * sizeof(double) );
    double * B = (double*) malloc( n * sizeof(double) );
    double * C = (double*) malloc( n * sizeof(double) );
    double * D = (double*) malloc( n * sizeof(double) );

    for (i = 0; i < n; ++i) {
        A[i] = ((double) rand()/RAND_MAX);
        B[i] = ((double) rand()/RAND_MAX);
    }

    // Add the two vectors in serial
    t0 = getTime();
    vecAdd(A, B, C, n);
    t1 = getTime();
    printf("\n  Serial addition: %f sec.", t1 - t0);

    // Add the two vectors using CUDA
    t2 = getTime();
    cuda_vecAdd(A, B, D, n);
    t3 = getTime();
    printf("\n  CUDA addition:   %f sec.\n\n", t3 - t2);

    // Verify that the two results are the same
    for (i = 0; i < n; ++i) {
        if( C[i] != D[i])
        {
            printf("\nERROR! Outputs do not match at index %d", i);
            break;
        }
    }

    // Free host memory
    free(A);
    free(B);
    free(C);
    free(D);

    return 0;
}
